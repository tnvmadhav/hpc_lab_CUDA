
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#define BLK_SIZE 16
using namespace std;

__global__ void gpuMM(double *a,double *b, double *c, int N)
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	double sum=0.0;
	if(row<N && col < N)
	{
		for(int i=0;i<N;i++)
			sum+=a[row*N+i]*b[i*N+col];
		c[row*N+col]=sum;
	}
	else
		return;
}

int main()
{
	int N,i,j,k;
	double *hA,*hB,*hC,*dA,*dB,*dC;

	cout<<"Enter N: ";
	cin>>N;


	hA = new double[N*N];
	hB = new double[N*N];
	hC = new double[N*N];
	int size = sizeof(double)*N*N;
	hipMalloc(&dA,size);
	hipMalloc(&dB,size);
	hipMalloc(&dC,size);


	for(i=0;i<N*N;i++)
	{
		hA[i] = i;
		hB[i] = N*N -1;
	}

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threads_per_block(BLK_SIZE,BLK_SIZE);
	dim3 no_of_blocks(ceil((float)N/BLK_SIZE),ceil((float)N/BLK_SIZE));

	hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
	hipMemcpy(dB,hB,size,hipMemcpyHostToDevice);

	
	hipEventRecord(start);	
	gpuMM<<<no_of_blocks,threads_per_block>>>(dA,dB,dC,N); /* function call for gpu action(gpuMM)(function call) */
	hipEventRecord(stop);

    hipMemcpy(hC,dC,size,hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);

	float milliseconds = 0.0;
	hipEventElapsedTime(&milliseconds,start,stop);
	

	double *cc = new double[N*N];
	double sum=0.0;

	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			sum=0.0;
			for(k=0;k<N;k++)
			{
				sum+= hA[i*N+k]*hB[k*N+j];
			}
			
			cc[i*N+j]=sum;
			if(hC[i*N+j] != cc[i*N+j])
			{
				cout<<"Incorrect Result\n";
				exit(0);
			}

		}
	}


	cout<<"Correct Result time: "<<milliseconds/1000<<endl;
	free(hA);free(hB);free(hC);free(cc);
	hipFree(dA);hipFree(dB);hipFree(dC);
}
